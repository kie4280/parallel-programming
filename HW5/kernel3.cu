#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#define threadsPerBlock 32
#define pixelPerThread 4

template <typename T>
void print(T a) {
  std::cout << a << std::endl;
}

__device__ int mandel(float c_re, float c_im, int maxIteration) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, int *buf, int resX,
                             int resY, float stepX, float stepY, size_t pitch,
                             int maxIterations) {
  // To avoid error caused by the floating number, use the following pseudo code
  //

  int rowX = blockIdx.x * blockDim.x + threadIdx.x;
  int rowY = blockIdx.y * blockDim.y + threadIdx.y;
  float y = lowerY + rowY * stepY;
  int *row = (int *)((char *)buf + rowY * pitch);
  int start = pixelPerThread * rowX;

  for (int a = 0; a < pixelPerThread; ++a) {
    int inx = (start + a);
    if (inx >= resX) break;
    float x = lowerX + inx * stepX;
    row[inx] = mandel(x, y, maxIterations);
  }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int *buf, *devMem;
  int thrPerRow = ((resX / pixelPerThread) + 1);
  int block_x = thrPerRow / threadsPerBlock + 1;
  int block_y = resY;

  size_t pitch;
  dim3 TB(threadsPerBlock, 1);
  dim3 GB(block_x, block_y);

  hipMallocPitch((void **)&devMem, &pitch, resX * sizeof(int), resY);
  mandelKernel<<<GB, TB>>>(lowerX, lowerY, devMem, resX, resY, stepX,
                           stepY, pitch, maxIterations);
  hipHostAlloc(&buf, resX * resY * sizeof(int), hipHostMallocDefault);
  hipMemcpy2D(buf, sizeof(int) * resX, devMem, pitch, resX * sizeof(int), resY,
               hipMemcpyDeviceToHost);
  // cudaDeviceSynchronize();

  for (int a = 0; a < resX * resY; ++a) {
    img[a] = buf[a];
  }

  hipFree(devMem);
  hipFree(buf);
  // free(buf);
}
