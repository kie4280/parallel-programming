#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#define blockWidth 8
#define blockHeight 8

template <typename T>
void print(T a) {
  std::cout << a << std::endl;
}

__device__ int mandel(float c_re, float c_im, int maxIteration) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float upperX, float upperY, float lowerX,
                             float lowerY, int* buf, int resX, int resY,
                             float stepX, float stepY, int maxIterations) {
  // To avoid error caused by the floating number, use the following pseudo code
  //

  int rowX = blockIdx.x * blockDim.x + threadIdx.x;
  int rowY = blockIdx.y * blockDim.y + threadIdx.y;

  float x = lowerX + rowX * stepX;
  float y = lowerY + rowY * stepY;
  // __syncthreads();
  if (rowX >= resX || rowY >= resY) {
    return;
  }
  buf[rowY * resX + rowX] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int block_x = resX / blockWidth + 1;
  int block_y = resY / blockHeight + 1;

  int* buf = (int*)malloc(resX * resY * sizeof(int));
  int* cudaMem;
  dim3 TB(blockWidth, blockHeight);
  dim3 BG(block_x, block_y);
  hipMalloc((void**)&cudaMem,
             block_x * block_y * blockWidth * blockHeight * sizeof(int));
  mandelKernel<<<BG, TB>>>(upperX, upperY, lowerX, lowerY, cudaMem, resX, resY,
                           stepX, stepY, maxIterations);
  hipMemcpy(buf, cudaMem, resX * resY * sizeof(int), hipMemcpyDeviceToHost);

  for (int a = 0; a < resX * resY; ++a) {
    img[a] = buf[a];
  }

  hipFree(cudaMem);
  free(buf);
}
