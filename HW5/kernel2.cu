#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

#define blockWidth 8
#define blockHeight 4

template <typename T>
void print(T a) {
  std::cout << a << std::endl;
}

__device__ int mandel(float c_re, float c_im, int maxIteration) {
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i) {
    if (z_re * z_re + z_im * z_im > 4.f) break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float upperX, float upperY, float lowerX,
                             float lowerY, int *buf, int resX, int resY,
                             float stepX, float stepY, size_t pitch,
                             int maxIterations) {
  // To avoid error caused by the floating number, use the following pseudo code
  //

  int rowX = blockIdx.x * blockDim.x + threadIdx.x;
  int rowY = blockIdx.y * blockDim.y + threadIdx.y;

  float x = lowerX + rowX * stepX;
  float y = lowerY + rowY * stepY;
  // __syncthreads();
  if (rowX >= resX || rowY >= resY) {
    return;
  }
  *((int *)((char *)buf + rowY * pitch) + rowX) = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  int block_x = resX / blockWidth + 1;
  int block_y = resY / blockHeight + 1;

  int *buf, *devMem;

  dim3 TB(blockWidth, blockHeight);
  dim3 GB(block_x, block_y);
  size_t pitch;
  hipMallocPitch((void **)&devMem, &pitch, resX * sizeof(int), resY);

  mandelKernel<<<GB, TB>>>(upperX, upperY, lowerX, lowerY, devMem, resX, resY,
                           stepX, stepY, pitch, maxIterations);
  hipHostAlloc(&buf, resX * resY * sizeof(int), hipHostMallocDefault);
  hipMemcpy2D(buf, sizeof(int) * resX, devMem, pitch, resX * sizeof(int), resY,
               hipMemcpyDeviceToHost);

  for (int a = 0; a < resX * resY; ++a) {
    img[a] = buf[a];
  }

  hipFree(devMem);
  hipFree(buf);
  // free(buf);
}
